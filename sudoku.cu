#include "hip/hip_runtime.h"

#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "util.h"

// The width and height of a sudoku board
#define BOARD_DIM 9

// The width and heigh of a square group in a sudoku board
#define GROUP_DIM 3

// The number of boards to pass to the solver at one time
#define BATCH_SIZE 2048

/**
 * A board is an array of 81 cells. Each cell is encoded as a 16-bit integer.
 * Read about this encoding in the documentation for the digit_to_cell and
 * cell_to_digit functions' documentation.
 *
 * Boards are stored as a one-dimensional array. It doesn't matter if you use
 * row-major or column-major form (that just corresponds to a rotation of the
 * sudoku board) but you will need to convert column and row to a single index
 * when accessing the board to propagate constraints.
 */
typedef struct board {
  uint16_t cells[BOARD_DIM * BOARD_DIM];
} board_t;

// Declare a few functions. Documentation is with the function definition.
void print_board(board_t* board);
__host__ __device__ uint16_t digit_to_cell(int digit);
__host__ __device__ int cell_to_digit(uint16_t cell);

/**
 * @brief This function updates a cells contraint set by checking the
 * contraints of other the cells in its column
 */
__device__ void check_col(board_t* board, int cell_ind, int* changed_constraints) {
  // Getting column index
  int col = threadIdx.x;

  // Index of last cell in the column
  int max = BOARD_DIM * (BOARD_DIM - 1) + col;

  // For the cells in that column
  for (int i = 0 * BOARD_DIM + col; i < max + 1; i += BOARD_DIM) {
    // If the cells are the same, skip this iteration
    if (i == cell_ind) {
      continue;
    }

    // If the the other cell in the column is a finite number
    if (cell_to_digit(board->cells[i]) != 0) {
      // Setting value of new cell
      uint16_t new_cell = board->cells[cell_ind] & ~(board->cells[i]);

      // If the bit in that position is not already zeroed out, then change
      if (board->cells[cell_ind] != new_cell) {
        board->cells[cell_ind] = new_cell;
        (*changed_constraints)++;
      }
    }
  }
}

/**
 * @brief This function updates a cells contraint set by checking the
 * contraints of other the cells in its row
 */
__device__ void check_row(board_t* board, int cell_ind, int* changed_constraints) {
  // Getting row index
  int row = threadIdx.y;

  // For the cells in that row
  for (int i = row * BOARD_DIM; i < (row + 1) * BOARD_DIM; i++) {
    // If the cells are the same, skip this iteration
    if (i == cell_ind) {
      continue;
    }

    // If the the other cell in the column is a finite number
    if (cell_to_digit(board->cells[i]) != 0) {
      // Setting value of new cell
      uint16_t new_cell = board->cells[cell_ind] & ~(board->cells[i]);

      // If the bit in that position is not already zeroed out, then change
      if (board->cells[cell_ind] != new_cell) {
        board->cells[cell_ind] = new_cell;
        (*changed_constraints)++;
      }
    }
  }
}

/**
 * @brief This function updates a cells contraint set
 * checking its section
 */
__device__ void check_sec(board_t* board, int cell_ind, int* changed_constraints) {
  // Setting start row and column index of the cell's
  // particular section
  int start_row = (threadIdx.y / 3) * 3;
  int start_col = (threadIdx.x / 3) * 3;

  // For the row
  for (int row = start_row; row < start_row + 3; row++) {
    // For the column
    for (int col = start_col; col < start_col + 3; col++) {
      // Get index of current cell
      int curr_cell = row * BOARD_DIM + col;

      // If the cells are the same, skip this iteration
      if (curr_cell == cell_ind) {
        continue;
      }

      // If the the other cell in the section is a finite number
      if (cell_to_digit(board->cells[curr_cell]) != 0) {
        // Setting value of new cell
        uint16_t new_cell = board->cells[cell_ind] & ~(board->cells[curr_cell]);

        // If the bit in that position is not already zeroed out, then change
        if (board->cells[cell_ind] != new_cell) {
          board->cells[cell_ind] = new_cell;
          (*changed_constraints)++;
        }
      }
    }
  }
}

/**
 * @brief Given a pointer to boards, this function solves the sudoku boards
 * that can be solved using the GPU, blocks and threads.
 *
 * @param boards a pointer to a list of board_t elements
 * @return __global__
 */
__global__ void solve_board_kernel(board_t* boards) {
  // Getting board to be solved
  board_t* board = &boards[blockIdx.x];

  // Getting cell index using the thread fields
  size_t cell_ind = threadIdx.y * BOARD_DIM + threadIdx.x;

  // Getting value of the cell in digits
  int16_t cell_val = board->cells[cell_ind];

  // Declaring changed_constraints to use later
  int changed_constraints;

  // If the current cell is not a finite number
  if (cell_to_digit(cell_val) == 0) {
    // Update the cells constraints.
    // Do this while any cell's contraints change.
    do {
      changed_constraints = 0;

      // Check column
      check_col(board, cell_ind, &changed_constraints);

      // Check row
      check_row(board, cell_ind, &changed_constraints);

      // Check section
      check_sec(board, cell_ind, &changed_constraints);

    } while (__syncthreads_count(changed_constraints) != 0);
  }
}

/**
 * Take an array of boards and solve them all. The number of boards will be no
 * more than BATCH_SIZE, but may be less if the total number of input boards
 * is not evenly-divisible by BATCH_SIZE.
 *
 * TODO: Implement this function! You will need to add a GPU kernel, and you
 *       will almost certainly want to write helper functions; that is fine.
 *       However, you should not modify any other functions in this file.
 *
 * \param boards      An array of boards that should be solved.
 * \param num_boards  The numebr of boards in the boards array
 */
void solve_boards(board_t* boards, size_t num_boards) {
  // Declaring pointer for gpu copy of boards.
  board_t* gpu_boards;

  // Allocate space for the gpu_boards array on the GPU
  if (hipMalloc(&gpu_boards, sizeof(board_t) * num_boards) != hipSuccess) {
    fprintf(stderr, "Failed to allocate X array on GPU\n");
    exit(2);
  }

  // Copy the boards array  from the cpu to the gpu
  if (hipMemcpy(gpu_boards, boards, sizeof(board_t) * num_boards, hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy Y from the CPU\n");
  }

  // Decalre dim3 variable for row and column used in kernel code
  dim3 thread_per_block(BOARD_DIM, BOARD_DIM);

  // Calling kernerl
  solve_board_kernel<<<num_boards, thread_per_block>>>(gpu_boards);

  // Wait for the kernel to finish
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
  }

  // Copy the boards array back from the gpu to the cpu
  if (hipMemcpy(boards, gpu_boards, sizeof(board_t) * num_boards, hipMemcpyDeviceToHost) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy Y from the GPU\n");
  }

  hipFree(gpu_boards);
}

/**
 * Take a digit from 0-9 (inclusive), and convert it to
 * cell form used for solving the sudoku. This encoding uses bits 1-9 to
 * indicate which values may appear in this cell.
 *
 * For example, if bit 3 is set to 1, then the cell may hold a three. Cells that
 * have multiple possible values will have multiple bits set.
 *
 * The input digit 0 is treated specially. This value indicates a blank cell,
 * where any value from one to nine is possible.
 *
 * \param digit   An integer value 0-9 inclusive
 * \returns       The encoded form of digit using bits to indicate which values
 *                may appear in this cell.
 */
__host__ __device__ uint16_t digit_to_cell(int digit) {
  if (digit == 0) {
    // A zero indicates a blank cell. Numbers 1-9 are possible, so set bits 1-9.
    return 0x3FE;
  } else {
    // Otherwise we have a fixed value. Set the corresponding bit in the board.
    return 1 << digit;
  }
}

/*
 * Convert an encoded cell back to its digit form. A cell with two or more
 * possible values will be encoded as a zero. Cells with one possible value
 * will be converted to that value.
 *
 * For example, if the provided cell has only bit three set, this function will
 * return the value 3.
 *
 * \param cell  An encoded cell that uses bits to indicate which values could
 *              appear at this point in the board.
 * \returns     The value that must appear in the cell if there is only one
 *              possibility, or zero otherwise.
 */
__host__ __device__ int cell_to_digit(uint16_t cell) {
  // Get the index of the least-significant bit in this cell's value
#if defined(__CUDA_ARCH__)
  int msb = __clz(cell);
  int lsb = sizeof(unsigned int) * 8 - msb - 1;
#else
  int lsb = __builtin_ctz(cell);
#endif

  // Is there only one possible value for this cell? If so, return it.
  // Otherwise return zero.
  if (cell == 1 << lsb)
    return lsb;
  else
    return 0;
}

/**
 * Read in a sudoku board from a string. Boards are represented as an array of
 * 81 16-bit integers. Each integer corresponds to a cell in the board. Bits
 * 1-9 of the integer indicate whether the values 1, 2, ..., 8, or 9 could
 * appear in the given cell. A zero in the input indicates a blank cell, where
 * any value could appear.
 *
 * \param output  The location where the board will be written
 * \param str     The input string that encodes the board
 * \returns       true if parsing succeeds, false otherwise
 */
bool read_board(board_t* output, const char* str) {
  for (int index = 0; index < BOARD_DIM * BOARD_DIM; index++) {
    if (str[index] < '0' || str[index] > '9') return false;

    // Convert the character value to an equivalent integer
    int value = str[index] - '0';

    // Set the value in the board
    output->cells[index] = digit_to_cell(value);
  }

  return true;
}

/**
 * Print a sudoku board. Any cell with a single possible value is printed. All
 * cells with two or more possible values are printed as blanks.
 *
 * \param board   The sudoku board to print
 */
void print_board(board_t* board) {
  for (int row = 0; row < BOARD_DIM; row++) {
    // Print horizontal dividers
    if (row != 0 && row % GROUP_DIM == 0) {
      for (int col = 0; col < BOARD_DIM * 2 + BOARD_DIM / GROUP_DIM; col++) {
        printf("-");
      }
      printf("\n");
    }

    for (int col = 0; col < BOARD_DIM; col++) {
      // Print vertical dividers
      if (col != 0 && col % GROUP_DIM == 0) printf("| ");

      // Compute the index of this cell in the board array
      int index = col + row * BOARD_DIM;

      // Get the index of the least-significant bit in this cell's value
      int digit = cell_to_digit(board->cells[index]);

      // Print the digit if it's not a zero. Otherwise print a blank.
      if (digit != 0)
        printf("%d ", digit);
      else
        printf("  ");
    }
    printf("\n");
  }
  printf("\n");
}

/**
 * Check through a batch of boards to see how many were solved correctly.
 *
 * \param boards        An array of (hopefully) solved boards
 * \param solutions     An array of solution boards
 * \param num_boards    The number of boards and solutions
 * \param solved_count  Output: A pointer to the count of solved boards.
 * \param error:count   Output: A pointer to the count of incorrect boards.
 */
void check_solutions(board_t* boards,
                     board_t* solutions,
                     size_t num_boards,
                     size_t* solved_count,
                     size_t* error_count) {
  // Loop over all the boards in this batch
  for (int i = 0; i < num_boards; i++) {
    // Does the board match the solution?
    if (memcmp(&boards[i], &solutions[i], sizeof(board_t)) == 0) {
      // Yes. Record a solved board
      (*solved_count)++;
    } else {
      // No. Make sure the board doesn't have any constraints that rule out
      // values that are supposed to appear in the solution.
      bool valid = true;
      for (int j = 0; j < BOARD_DIM * BOARD_DIM; j++) {
        if ((boards[i].cells[j] & solutions[i].cells[j]) == 0) {
          valid = false;
        }
      }

      // If the board contains an incorrect constraint, record an error
      if (!valid) (*error_count)++;
    }
  }
}

/**
 * Entry point for the program
 */
int main(int argc, char** argv) {
  // Check arguments
  if (argc != 2) {
    fprintf(stderr, "Usage: %s <input file name>\n", argv[0]);
    exit(1);
  }

  // Try to open the input file
  FILE* input = fopen(argv[1], "r");
  if (input == NULL) {
    fprintf(stderr, "Failed to open input file %s.\n", argv[1]);
    perror(NULL);
    exit(2);
  }

  // Keep track of total boards, boards solved, and incorrect outputs
  size_t board_count = 0;
  size_t solved_count = 0;
  size_t error_count = 0;

  // Keep track of time spent solving
  size_t solving_time = 0;

  // Reserve space for a batch of boards and solutions
  board_t boards[BATCH_SIZE];
  board_t solutions[BATCH_SIZE];

  // Keep track of how many boards we've read in this batch
  size_t batch_count = 0;

  // Read the input file line-by-line
  char* line = NULL;
  size_t line_capacity = 0;
  while (getline(&line, &line_capacity, input) > 0) {
    // Read in the starting board
    if (!read_board(&boards[batch_count], line)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Read in the solution board
    if (!read_board(&solutions[batch_count], line + BOARD_DIM * BOARD_DIM + 1)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Move to the next index in the batch
    batch_count++;

    // Also increment the total count of boards
    board_count++;

    // If we finished a batch, run the solver
    if (batch_count == BATCH_SIZE) {
      size_t start_time = time_ms();
      solve_boards(boards, batch_count);
      solving_time += time_ms() - start_time;

      check_solutions(boards, solutions, batch_count, &solved_count, &error_count);

      // Reset the batch count
      batch_count = 0;
    }
  }

  // Check if there's an incomplete batch to solve
  if (batch_count > 0) {
    size_t start_time = time_ms();
    solve_boards(boards, batch_count);
    solving_time += time_ms() - start_time;

    check_solutions(boards, solutions, batch_count, &solved_count, &error_count);
  }

  // Print stats
  double seconds = (double)solving_time / 1000;
  double solving_rate = (double)solved_count / seconds;

  // Don't print nan when solver is not implemented
  if (seconds < 0.01) solving_rate = 0;

  printf("Boards: %lu\n", board_count);
  printf("Boards Solved: %lu\n", solved_count);
  printf("Errors: %lu\n", error_count);
  printf("Total Solving Time: %lums\n", solving_time);
  printf("Solving Rate: %.2f sudoku/second\n", solving_rate);

  return 0;
}